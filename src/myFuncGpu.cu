#include "hip/hip_runtime.h"
#include "myFuncOmp.h"
#include <hip/hip_runtime.h>



//allocate memory for string
int cudaAllocateString(char **str,int str_len){
    hipError_t cudaStatus;
    *str = 0;
    cudaStatus = hipMalloc((void**)str,str_len*sizeof(char));
    //if there is an error then we will get into the if statement
    if(cudaStatus != hipSuccess){
        printf("Failed allocating cuda\n");
        goto error;
    }
    return 1;
error:
    hipFree(*str);
    return 0;

}
//allocate memory for int
int cudaAllocateIntArray(int ** ptr,int num_elemnts)
{
    hipError_t cudaStatus;
    *ptr = 0;
    cudaStatus = hipMalloc((void**)ptr,num_elemnts*sizeof(int));
    //if there is an error then we will get into the if statement
    if(cudaStatus != hipSuccess){
        printf("Failed allocating cuda\n");
        goto error;
    }
    return 1;
error:
    hipFree(*ptr);
    return 0;

}

//copy to the gpu
int cudaCopyToDevice(char* cpu_mem,char* gpu_mem,int length){
    hipError_t cudaStatus;
    cudaStatus = hipMemcpy(gpu_mem,cpu_mem,length*sizeof(char),hipMemcpyHostToDevice); 
    //if there is an error then we will get into the if statement 
    if(cudaStatus != hipSuccess){
        printf("Failed copy to device cuda\n");
        return 0;
    }
    return 1;

}

//copy to the cpu
int cudaCopyToHost(char* cpu_mem,char* gpu_mem,int length){
    hipError_t cudaStatus;
    cudaStatus = hipMemcpy(cpu_mem,gpu_mem,length*sizeof(char),hipMemcpyDeviceToHost); 
    //if there is an error then we will get into the if statement   
    if(cudaStatus != hipSuccess){
        printf("Failed copy to host cuda\n");
        return 0;
    }
    return 1;
}

int cudaCopyIntToHost(int* cpu_mem,int* gpu_mem,int length){
    hipError_t cudaStatus;
    cudaStatus = hipMemcpy(cpu_mem,gpu_mem,length*sizeof(int),hipMemcpyDeviceToHost); 
    //if there is an error then we will get into the if statement    
    if(cudaStatus != hipSuccess){
        printf("Failed copy to host cuda\n");
        return 0;
    }
    return 1;
}
//cuda free allocation
void cudaFreeFromHost(void* ptr)
{
    hipFree(ptr);
}

__device__ int strLenCuda(char* str){
    int length = 0;
    while(*str != 0){
        length++;
        str++;
    }
    return length;
}

__device__ bool checkIfMatchCuda(unsigned char ch1,unsigned char ch2){
    return ch1 == ch2? true : false;
}

__device__ bool checkWordAndCipherCuda(int input_length, unsigned char *possible_plaintext_str, char *word)
{
    //initalization
    int i = 0;
    int k = 0;
    int numOfSuccesses = 0;
    int wordLength = strLenCuda(word);
    int in_word = 0;
    //same algorithm as myFuncOmp but here we calculate in the gpu.
    for (i=0; i < input_length; i++)
    {
        if (k < wordLength)
        {
            if (possible_plaintext_str[i] == ' ')
            {
                in_word = 0;
                continue;
            }
            if (in_word == 0 || k > 0)
            {
                if (checkIfMatchCuda((unsigned char)word[k], possible_plaintext_str[i]))
                {
                    numOfSuccesses += 1;
                    k++;
                }
                else
                {
                    k = 0;
                    numOfSuccesses = 0;
                }
            }
            in_word = 1;
        }
        else
        {
            if (possible_plaintext_str[i] != ' ')
            {
                k = 0;
                numOfSuccesses = 0;
            }
            else
            {
                break;
            }
        }
    }

    return numOfSuccesses == wordLength ? true : false;
}


__global__ void kernelXor(unsigned int key,char* input_str_cuda,unsigned char* possible_plaintext_str_cuda,int input_length){
    //initalization
    int id = threadIdx.x + blockDim.x*blockIdx.x;
    //if the id is the same ar the input length then we return.
    if(id >= input_length)
        return;
    int keyIndex = id%4;
    char* keyCharPtr = ((char*)&key);
    char keyChar = keyCharPtr[keyIndex];
    //calculate the xor and save the result at the possible plaintext array of cuda.
    possible_plaintext_str_cuda[id] = keyChar ^ input_str_cuda[id];
}

__global__ void kernelFindWords(unsigned char* possible_plaintext_str_cuda,int input_length,char* known_words_array_cuda,int known_words_length,int* cuda_results,int resIndex,int pow_of_2){
    //initalization
    int id = threadIdx.x + blockDim.x*blockIdx.x;
    unsigned int s;
    int sum;
    //we need the shared memory for gathering all the results from each block.
    __shared__ int wordsFound[1024];
    if(id >= known_words_length)
        return;
    wordsFound[threadIdx.x]=0;
    char* myWord = known_words_array_cuda+id*MAX_STRING_LENGTH;
    //if we find somthing we want to increase with 1 at the index of the threadId
    if(checkWordAndCipherCuda(input_length,possible_plaintext_str_cuda,myWord)){
        wordsFound[threadIdx.x]=1;
    }
    __syncthreads();
	//here i take care of extreme case - if the number of block isn't pow of 2(if i dont take care of this case then it will not work at all).
    if(pow_of_2/2 != blockDim.x)
    {
        s = pow_of_2/2;
        if (threadIdx.x < s) 
        { 
            if(threadIdx.x + s < blockDim.x)
                wordsFound[threadIdx.x] += wordsFound[threadIdx.x + s];
        }
        __syncthreads();
    }
    else
        s = blockDim.x;
    for(s=s/2; s>0; s=s/2) 
    {
        if (threadIdx.x < s) 
        {
            wordsFound[threadIdx.x] += wordsFound[threadIdx.x + s];
        }
        __syncthreads();
    }
    //kind of reduction i want to amount all the result into cuda_results at the res index(that we send from myFunOmp.c) .
    if(threadIdx.x == 0)
    {
        sum = wordsFound[0];
        atomicAdd(&cuda_results[resIndex],sum);
    }
    __syncthreads();
    
    
    
}

int nextPowerOf2( int n)  
{  
   	int count = 0;  
      
    // First n in the below condition  
    // is for the case where n is 0  
    if (n && !(n & (n - 1)))  
        return n;  
      
    while( n != 0)  
    {  
        n >>= 1;  
        count += 1;  
    }  
      
    return 1 << count;  
}




void gpuDecryption(unsigned int key, char* known_words_array_cuda,char* input_str_cuda,unsigned char* possible_plaintext_str_cuda,int* cuda_results,int known_words_length,int input_length,int resIndex){
    //initalization
    int numThreads;
    int numBlocks;
    int extraBlock;
    
    hipError_t cudaStatus;
    hipDeviceProp_t props;

    //Properties for the specified device
    hipGetDeviceProperties(&props,0);

    //calculate num threads needed according to input length
    numThreads = props.maxThreadsPerBlock < input_length ? props.maxThreadsPerBlock : input_length;
    numBlocks = input_length/numThreads;
    extraBlock = input_length%numThreads != 0;

    //sending the paramters with the for calculate the xor with the possible key that we get from the function that call us (calculatePossibleKey at myFuncOmp.c)
    kernelXor<<<numBlocks+extraBlock,numThreads>>>(key,input_str_cuda,possible_plaintext_str_cuda,input_length);

    //wait for the kernel
    cudaStatus = hipDeviceSynchronize();

    //if there is any problem i print it into the terminal.
    if(cudaStatus != hipSuccess){
        printf("Error in kernelxor\n");
        return ;
    }

    //calculate the number of threads needed for finding the words same way as i calculate for the input length.
    numThreads = props.maxThreadsPerBlock < known_words_length ? props.maxThreadsPerBlock : known_words_length;

    //calculate the number of block needed.
    numBlocks = known_words_length/numThreads;
    extraBlock = known_words_length%numThreads != 0;
    int pow_of_2 = nextPowerOf2(numThreads);
    //calling the find words at the known words array of cuda.
    kernelFindWords<<<numBlocks+extraBlock,numThreads>>>(possible_plaintext_str_cuda,input_length,known_words_array_cuda,known_words_length,cuda_results,resIndex,pow_of_2*2);

    //wait for the kernel
    cudaStatus = hipDeviceSynchronize();
    if(cudaStatus != hipSuccess){
        printf("Error in kernelFindWords\n");
        return ;
    }
}


